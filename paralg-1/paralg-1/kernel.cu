
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>

int main() {
	thrust::host_vector<int> data_h;
	thrust::device_vector<int> data_d;
	thrust::device_vector<int> out_d;

	for (int i = 0; i < 10; i++) {
		int x = i + ((i % 3) == 0) * 2;
		data_h.push_back(x);
		printf("%3d ", x);
	}
	putchar('\n');

	data_d = data_h;
	out_d.resize(data_d.size());

	thrust::exclusive_scan(data_d.begin(), data_d.end(), out_d.begin(), -1, thrust::maximum<int>());

	data_h = out_d;
	for (thrust::host_vector<int>::iterator i = data_h.begin(); i != data_h.end(); i++)
		printf("%3d ", *i);
	putchar('\n');
}