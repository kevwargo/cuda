#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>

struct bitCount : public thrust::binary_function<int, int, int> {
	// doesn't work because it's not associative
	__device__ int operator()(int x, int y) {
		return x + __popc(y);
	}
};

int main() {
	thrust::host_vector<int> src_h;
	thrust::device_vector<int> src_d;

	for (int i = 0; i < 10; i++) {
		src_h.push_back(i);
	}

	src_d = src_h;

	int bit_sum = thrust::reduce(src_d.begin(), src_d.end(), 0, bitCount());

	printf("%d\n", bit_sum);
}