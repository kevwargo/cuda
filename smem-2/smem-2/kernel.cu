#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <stdio.h>
#include <ctime>


__global__ void movingAverage(int size, int d, int *data)
{
	extern __shared__ int smem[];
	int btid = threadIdx.x;
	int gtid = btid + blockIdx.x*blockDim.x;
	int idx1 = gtid - (d - 1) / 2;
	int idx2 = blockDim.x + gtid - (d - 1) / 2;
	
	if (idx1 > size) {
		return;
	}
	if (idx2 > size) {
		return;
	}

	smem[btid] = data[idx1];
	if (btid < d - 1) {
		smem[blockDim.x + btid] = data[idx2];
	}

}

int main()
{
	int bs = 512;
	int gs = 32;
	int size = bs * gs;
	int d = 7;
	thrust::host_vector<int> data_h;
	thrust::device_vector<int> data_d;
	int *movavg = new int[size];
	srand(time(NULL));

	for (int i = 0; i < size; i++) {
		int x = rand() % 100;
		data_h.push_back(x);
	}
	for (int i = 0; i < size; i++) {
		int sum = 0;
		int start = i - (d - 1) / 2;
		int end = i + (d - 1) / 2;
		while (start < 0) {
			sum += data_h[0];
			start++;
		}
		while (end >= size) {
			sum += data_h[size - 1];
			end--;
		}

		for (int j = start; j <= end; j++) {
			sum += data_h[j];
		}
		movavg[i] = sum / d;
	}
	
	data_d = data_h;
	movingAverage << <gs, bs, sizeof(int)*(bs + d - 1) >> > (size, d, data_d.data().get());
	data_h = data_d;

	int errcount = 0;
	for (int i = 0; i < size; i++) {
		if (data_h[i] != movavg[i]) {
			errcount++;
		}
	}

	printf("Error count: %d\n", errcount);
}
